#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/BinaryOps.h>

// TODO: update to use lazynvrtc
#include <ATen/cuda/detail/LazyNVRTC.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>
#include <torch/csrc/jit/resource_guard.h>
#include <sstream>
#include <torch/csrc/jit/frontend/code_template.h>
#include <torch/csrc/jit/codegen/fuser/cuda/fused_kernel.h>
#include <algorithm>
#include <cctype>
#include <unordered_map>
#include <c10/core/ScalarType.h>
#include <c10/util/Optional.h>
#include <mutex>

namespace at { namespace native {
namespace {

// TODO jiterator cache design does not handle multiple gpus currently
using JiteratorKey = int32_t;
using JiteratorCache = std::unordered_map<JiteratorKey, hipFunction_t>;

// global jiterator mutex
// TODO: currently caches are per function but the mutex is global,
//   so maybe mutexes should be per function, too, or the caches should
//   be consolidated
std::mutex jiterator_mutex;

JiteratorKey construct_jiterator_key(const ScalarType scalar_type, bool dynamic_casting) {
  return (uint8_t)scalar_type * 2 + dynamic_casting;
}

// NOTE: get does not acquire the lock
c10::optional<hipFunction_t> get_jitted_function(const JiteratorCache& cache, JiteratorKey key) {
  auto it = cache.find(key);
  if (it == cache.end()) {
    return c10::nullopt;
  }
  return it->second;
}

static inline std::string string_repr(ScalarType t) {
#define CASE_STRING_CASE(ctype, name) \
  case ScalarType::name:                   \
    return std::string(#ctype);

  switch (t) {
    //TODO use a different macro that starts with commonly used types
    AT_FORALL_SCALAR_TYPES_WITH_COMPLEX_AND_QINTS(CASE_STRING_CASE)
    default:
      AT_ERROR("Unknown ScalarType");
  }
#undef CASE_STRING_CASE
}

void store_jitted_function(
    JiteratorCache& cache,
    const JiteratorKey key,
    hipFunction_t function) {
  cache.emplace(key, function);
}

constexpr int num_threads = 64;
constexpr int thread_work_size = 4; //TODO make template substitution once we decide where those vars live
constexpr int block_work_size = thread_work_size * num_threads;

#define stringify(...) std::string(#__VA_ARGS__); __VA_ARGS__
const auto jittable_foo_functor = stringify(
  template<typename scalar_t>
  struct FooFunctor {
    FooFunctor(scalar_t a): alpha{a} {}
    __device__ __forceinline__ scalar_t operator() (const scalar_t a, const scalar_t b) const {
      return a + alpha * b;
    }

    scalar_t alpha;
  };
);
#undef stringify

std::string generate_code(
    const TensorIterator& iter,
    bool dynamic_casting) {
  // Constructs kernel code
  const int nInputs = iter.ninputs();
  torch::jit::TemplateEnv env;
  env.s("name", "FooFunctor");
  env.s("functor", jittable_foo_functor);
  env.s("index_type", "unsigned int");
  env.s("nInputs", std::to_string(nInputs));
  // Identifies scalar type
  // TODO: there has to be an existing way of doing this (i.e. converting scalar type to string)
  const auto& common_dtype = iter.common_dtype();
  std::string common_dtype_string = string_repr(common_dtype);
  // if (common_dtype == kFloat) {
  //   common_dtype_string = "float";
  // } else if (common_dtype == kDouble) {
  //   common_dtype_string = "double";
  // }
  env.s("scalar_type", common_dtype_string);
  std::stringstream declare_load_arrays;
  for (int i=0; i < nInputs; i++){
//TODO these arrays are potentially of the different types, use function traits to determine the types
    declare_load_arrays << common_dtype_string << " arg" << std::to_string(i) << "[" << std::to_string(thread_work_size) << "];\n";
  }
  env.s("declare_load_arrays", declare_load_arrays.str());
  std::stringstream declare_store_arrays;
  declare_store_arrays << common_dtype_string << " out" << "[" << std::to_string(thread_work_size) << "];\n";
  env.s("declare_store_arrays", declare_store_arrays.str());
  if (!dynamic_casting) {
    env.s("loader", "LoadWithoutCast");
    env.s("storer", "StoreWithoutCast");
  } else {
    env.s("loader", std::string("LoadWithCast<"+std::to_string(nInputs) + ">"));
    env.s("storer", "StoreWithCast");
  }
  std::stringstream load_inputs;
  for (int i=0; i < nInputs; i++){
    auto i_string = std::to_string(i);
    load_inputs << "arg" << i_string << "[j] = l.load<"
                << common_dtype_string << ">(data["
                << std::to_string(i + iter.noutputs()) << "], input_offsets["
                << i_string << "], " << i_string << ");\n";
  }
  env.s("load_inputs", load_inputs.str());
  std::stringstream store_outputs;
  store_outputs << "s.store<" << common_dtype_string
                << ">(out[j], data[0], output_offsets[0]);\n";
  env.s("store_outputs", store_outputs.str());
  std::stringstream functor_args;
  for (int i=0; i < nInputs - 1; i++){
    functor_args << "arg" << std::to_string(i) << "[j], ";
  }
  functor_args << "arg" << std::to_string(nInputs-1) << "[j]";
  env.s("args", functor_args.str());
  static auto cuda_template = at::cuda::detail::load_code_template("/private/home/ngimel/pytorch/aten/src/ATen/native/cuda/code_template.cuh");
  return  cuda_template.format(env);
}

hipFunction_t jit_pwise_function(
    JiteratorCache& cache,
    JiteratorKey key,
    const std::string& code,
    const std::string& kernel_name) {

  // TODO: this lock is could be acquired around the cache updates
  std::lock_guard<std::mutex> guard{jiterator_mutex};

  // Compiles the kernel ---

  // Acquires device and NVRTC properties (for compile arch and occupancy calculations)
  hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
  int major, minor;
  torch::jit::fuser::cuda::getMajorMinor(prop, major, minor);

  // Creates the NVRTC program
  hiprtcProgram program;
  const auto& nvrtc = at::globalContext().getNVRTC();
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcCreateProgram(
      &program, code.c_str(), nullptr, 0, nullptr, nullptr));
  // constructs nvrtc build arguments
  const std::string compute = "--gpu-architecture=compute_" +
    std::to_string(major) + std::to_string(minor);
  const std::vector<const char*> build_args = {
    "--std=c++14", compute.c_str(), "-default-device"};

  // compiles and validates result
  const auto compilation_result =
        nvrtc.hiprtcCompileProgram(program, build_args.size(), build_args.data());
  if (compilation_result != HIPRTC_SUCCESS) {
    size_t logsize;
    AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetProgramLogSize(program, &logsize));
    std::vector<char> log(logsize);
    AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetProgramLog(program, log.data()));
    std::stringstream cu;
    cu << log.data();
    throw std::runtime_error(cu.str());
  }

  hipModule_t module;
  hipFunction_t function;
  std::vector<char> ptx;
  size_t ptx_size;
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetCodeSize(program, &ptx_size));
  ptx.resize(ptx_size);
  AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcGetCode(program, ptx.data()));
  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleLoadData(&module, ptx.data()));
  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleGetFunction(&function, module, kernel_name.c_str()));


  // Updates (or not) the cache and returns the function ---
  c10::optional<hipFunction_t> maybe_function = get_jitted_function(cache, key);
  if (maybe_function) {
    // Destroys the just compiled but unneccessary program
    AT_CUDA_NVRTC_CHECK(nvrtc.hiprtcDestroyProgram(&program));
    return *maybe_function;
  }

  store_jitted_function(cache, key, function);
  return function;
}

// TODO: may need/want to initialize CUDA context here (refactor into nvrtc call)
void launch_jitted_pwise_function(
    hipFunction_t function,
    std::vector<void*>& args,
    const int nBlocks,
    const int kBlockSize) {

  const auto& nvrtc = at::globalContext().getNVRTC();

  // TODO: seems like this and block calculation should be cached per device
  // Acquires device and NVRTC properties (for compile arch and occupancy calculations)
  hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
  int major, minor;


  // Launches kernel on current stream
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_CUDA_DRIVER_CHECK(nvrtc.hipModuleLaunchKernel(
    function,
    nBlocks,
    1,
    1,
    kBlockSize,
    1,
    1,
    0,
    stream,
    args.data(),
    nullptr));
}

//launch has to happen in this function because of lifetime of
//objects going into args vector
template <typename func_t>
void prepare_args_and_launch_impl(hipFunction_t function, TensorIterator iter, func_t f,
bool needs_dynamic_cast){
  constexpr int nargs = function_traits<func_t>::arity;
// Constructs kernel args
  std::vector<void*> args;
  args.push_back((void*)&f);
  // Adds numel arg
  // NOTE: the intermediate capture is neccessary
  const int64_t numel = iter.numel();
  args.push_back((void*)&numel);

  // Adds data ptrs
  at::detail::Array<char*, nargs+1> data;
  for (auto i = decltype(iter.ntensors()){0}; i < iter.ntensors(); i++) {
    data[i] = (char*)iter.data_ptr(i);
  }
  args.push_back((void*)&data);

  // Addds offset calculators
  // TODO: maybe combine into one offset calculator?
  auto input_offset_calculator = make_input_offset_calculator<nargs>(iter);
  auto output_offset_calculator = make_output_offset_calculator(iter);
  args.push_back((void*)&input_offset_calculator);
  args.push_back((void*)&output_offset_calculator);

  int64_t grid = (numel + block_work_size - 1) / block_work_size;
  if (needs_dynamic_cast) {
    at::detail::Array<ScalarType, nargs> dtypes;
    for (int i = 0; i < iter.ninputs(); i++) {
      dtypes[i] = iter.tensor(i + iter.noutputs()).scalar_type();
    }
    auto loader = memory::LoadWithCast<nargs>(dtypes);
    auto storer = memory::StoreWithCast(iter.tensor(0).scalar_type());
    args.push_back((void*)&loader);
    args.push_back((void*)&storer);
    launch_jitted_pwise_function(function, args, grid, num_threads);
    // TORCH_CHECK(false, "dynamic cast not supported yet")
  } else {
    auto loader = memory::LoadWithoutCast();
    auto storer = memory::StoreWithoutCast();
    args.push_back((void*)&loader);
    args.push_back((void*)&storer);
    // need to launch inside the if block because of loader runtime
    // alternative is to make this function templated on loader and storer types
    launch_jitted_pwise_function(function, args, grid, num_threads);
  }
}

template <typename func_t>
void prepare_args_and_launch(hipFunction_t function, TensorIterator iter, func_t f, bool needs_dynamic_cast){
  if (iter.numel() == 0) {
    return;
  }
  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      prepare_args_and_launch(function, sub_iter, f, needs_dynamic_cast);
    }
    return;
  }
  prepare_args_and_launch_impl(function, iter, f, needs_dynamic_cast);
}

} // anonymous namespace



JiteratorCache foo_cache;

Tensor foo_cuda(const Tensor& self, const Tensor& other, Scalar alpha_scalar) {
  Tensor result;
  auto iter = TensorIterator::binary_op(result, self, other);

  TORCH_INTERNAL_ASSERT(iter.ntensors() == 3);

  std::cout << "dtype 0: " << iter.dtype(0) << std::endl;
  std::cout << "dtype 1: " << iter.dtype(1) << std::endl;
  std::cout << "dtype 2: " << iter.dtype(2) << std::endl;
  std::cout << "iter.tensor(0).scalar_type(): " << iter.tensor(0).scalar_type() << std::endl;
  std::cout << "iter.tensor(1).scalar_type(): " << iter.tensor(1).scalar_type() << std::endl;
  std::cout << "iter.tensor(2).scalar_type(): " << iter.tensor(2).scalar_type() << std::endl;
  std::cout << "common_dtype: " << iter.common_dtype() << std::endl;
  // std::cout << "jittable functor string" << std::endl;
  // std::cout << jittable_foo_functor << std::endl;

  // Creates functor arg
  // TODO: refactor with dispatch macro?
  // TODO: support float or double dynamically


  AT_DISPATCH_ALL_TYPES(iter.common_dtype(), "foo", [&]() {
    FooFunctor<scalar_t> my_functor{alpha_scalar.to<scalar_t>()};
    bool dynamic_casting = needs_dynamic_casting<FooFunctor<scalar_t>>::check(iter);

    JiteratorKey key = construct_jiterator_key(iter.common_dtype(), dynamic_casting);
    c10::optional<hipFunction_t> maybe_function = get_jitted_function(foo_cache, key);
    hipFunction_t function;
    if (maybe_function) {
      std::cout << "found function" << std::endl;
      function = *maybe_function;
    } else {
      std::cout << "jitting function" << std::endl;
      // TODO: make kernel name generic
      // Note: even though code is generated on an iter that can potentially
      // be split, the properties of the iter that are used for codegen
      // won't change if it is split
      auto code = generate_code(iter, dynamic_casting);
  //    std::cout << "code " << code << "\n";
      const std::string kernel_name{"FooFunctor_kernel"};
      function = jit_pwise_function(foo_cache, key, code, kernel_name);
    }
    prepare_args_and_launch(function, iter, my_functor, dynamic_casting);
  });
  return iter.output();
}

}} // namespace at::native
