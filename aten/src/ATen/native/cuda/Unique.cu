#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>

#include <tuple>
#include <iterator>
#include <thrust/adjacent_difference.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/iterator/zip_iterator.h>

namespace at {
namespace native{

namespace {


template <
  typename policy_t, typename scalar_t,
  typename equal_t, typename not_equal_t
>
std::tuple<Tensor, Tensor, Tensor, int64_t> compute_unique(
  const policy_t &policy,
  scalar_t *data,
  int64_t num_inp,
  const Tensor &sorted_indices,
  const bool return_inverse,
  const bool return_counts,
  const bool return_indices,
  TensorOptions options,
  equal_t equal,
  not_equal_t not_equal
) {

  // inverse indices
  if (return_inverse || return_indices) {
    TORCH_CHECK(sorted_indices.defined(),
    "return_inverse or return_indices is set to true, but sorted_indices is undefined. Send a bug report!");
  }
  Tensor inverse_indices;
  if (!return_inverse || num_inp == 0) {
    inverse_indices = at::empty({0}, options);
  } else {
    const int64_t *sorted_indices_ptr = sorted_indices.data_ptr<int64_t>();
    Tensor inv_loc = at::empty({num_inp}, options);
    inverse_indices = at::empty({num_inp}, options);
    int64_t* inv_loc_ptr = inv_loc.data_ptr<int64_t>();
    int64_t* inverse_indices_ptr = inverse_indices.data_ptr<int64_t>();
    thrust::adjacent_difference(policy, data, data + num_inp, inv_loc_ptr, not_equal);
    inv_loc[0] = 0;
    thrust::inclusive_scan(policy, inv_loc_ptr, inv_loc_ptr + num_inp, inv_loc_ptr);
    thrust::scatter(policy, inv_loc_ptr, inv_loc_ptr + num_inp, sorted_indices_ptr, inverse_indices_ptr);
  }

  // unique and count
  Tensor counts = at::empty({0}, options);
  Tensor indices = at::empty({0}, options);
  int64_t num_out;
  if (!return_counts && !return_indices) {
    num_out = thrust::unique(policy, data, data + num_inp, equal) - data;
  } else {
    if (return_counts) {
      Tensor range = at::arange(0, num_inp + 1, options);
      int64_t *range_ptr = range.data_ptr<int64_t>();
      if (!return_indices) {
        num_out = thrust::unique_by_key(policy, data, data + num_inp, range_ptr, equal).first - data;
      } else { //both r_c and r_i requested
        indices = at::empty({num_inp}, options).copy_(sorted_indices); // cannot operate directly on sorted indices, because
        //for unique_dim they are also used in unique key
        int64_t *indices_ptr = indices.data_ptr<int64_t>();
        num_out = thrust::unique_by_key(policy, data, data + num_inp,
        thrust::make_zip_iterator(thrust::make_tuple(range_ptr, indices_ptr)), equal).first - data;
        indices.resize_(num_out);
      }
      range[num_out] = num_inp;
      counts.resize_(num_out);
      int64_t* counts_ptr = counts.data_ptr<int64_t>();
      thrust::adjacent_difference(policy, range_ptr + 1, range_ptr + num_out + 1, counts_ptr);
    } else if (return_indices) { //r_c=false, r_i = true
      //we don't know exact number of future unique items, so unfortunately will have to allocate full size indices
      indices = at::empty({num_inp}, options).copy_(sorted_indices); // cannot operate directly on sorted indices, because
      //for unique_dim they are also used in unique key
      int64_t *indices_ptr = indices.data_ptr<int64_t>();
      num_out = thrust::unique_by_key(policy, data, data + num_inp, indices_ptr, equal).first - data;
      indices.resize_(num_out);
    }



  }


  AT_CUDA_CHECK(hipGetLastError());
  return std::tuple<Tensor, Tensor, Tensor, int64_t>(inverse_indices, counts, indices, num_out);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor, Tensor> unique_cuda_template(
  const Tensor& self,
  const bool consecutive,
  const bool return_inverse,
  const bool return_counts,
  const bool return_indices
) {

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto options = self.options().dtype(kLong);
  Tensor output = self.clone(at::MemoryFormat::Contiguous).reshape(-1);
  int64_t num_inp = output.numel();
  scalar_t* output_data = output.data_ptr<scalar_t>();

  Tensor sorted_indices;
  if (!return_inverse && !return_indices) {
    if (!consecutive) {
      thrust::sort(policy, output_data, output_data + num_inp);
    }
  } else {
    sorted_indices = at::arange(0, num_inp, options);
    if (!consecutive) {
      int64_t *sorted_indices_ptr = sorted_indices.data_ptr<int64_t>();
      thrust::sort_by_key(policy, output_data, output_data + num_inp, sorted_indices_ptr);
    }
  }

  Tensor inverse_indices, counts, indices;
  int64_t num_out;
  std::tie(inverse_indices, counts, indices, num_out) = compute_unique(
    policy, output_data, num_inp, sorted_indices,
    return_inverse, return_counts, return_indices, options,
    thrust::equal_to<scalar_t>(),
    thrust::not_equal_to<scalar_t>()
  );
  output.resize_(num_out);

  if (return_inverse) {
      inverse_indices.resize_(self.sizes());
  }

  return std::tuple<Tensor, Tensor, Tensor, Tensor>(output, inverse_indices, counts, indices);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor, Tensor> unique_dim_cuda_template(
  const Tensor& self,
  const int64_t dim,
  const bool consecutive,
  const bool return_inverse,
  const bool return_counts,
  const bool return_indices
) {

  /**
    * The idea for implementing this is basically the same as unique.
    * For unique_dim, we are taking the unique with respect to a index
    * tensor, but during the processes, we override the compare and equal
    * operator by checking the data underlying it instead. After the
    * algorithm, we would use index_select to map the resulting indicies
    * to the result on the actual data.
    */

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto sizes = self.sizes().vec();
  // check how many zero dimensions exist
  auto num_zero_dims = std::count(sizes.begin(), sizes.end(), 0);

  // tensor is not well formed as it has 0 sized dimensions
  if (self.size(dim) == 0){
    TORCH_CHECK(
        num_zero_dims == 1,
        "Number of zero sized dimensions is more than one, so unique cannot be applied ")
    Tensor output = at::empty({0}, self.options());
    Tensor inverse_indices =
        at::empty({0}, self.options().dtype(kLong));
    Tensor counts = at::empty({0}, self.options().dtype(kLong));
    Tensor indices = at::empty({0}, self.options().dtype(kLong));
    return std::make_tuple(output, inverse_indices, counts, indices);
  }

  TORCH_CHECK(num_zero_dims == 0,
    "There are 0 sized dimensions, and they aren't selected, so unique cannot be applied");

  int64_t num_inp = self.size(dim);
  auto options = self.options().dtype(kLong);
  Tensor input_flat = self.transpose(dim, 0).contiguous().view({num_inp, -1});
  int64_t n = input_flat.size(1);
  scalar_t *input_flat_ptr = input_flat.data_ptr<scalar_t>();

  Tensor indices = at::arange(0, num_inp, options);
  int64_t *indices_data = indices.data_ptr<int64_t>();
  if (!consecutive) {
    thrust::sort(policy, indices_data, indices_data + num_inp,
      [=] __device__ (int64_t a, int64_t b) -> bool {
        for (int64_t i = 0; i < n; ++i) {
          scalar_t lhs = input_flat_ptr[i + a * n];
          scalar_t rhs = input_flat_ptr[i + b * n];
          if (lhs < rhs) {
            return true;
          } else if (lhs > rhs) {
            return false;
          }
        }
        return false;
      }
    );
  }

  Tensor inverse_indices, counts, unique_indices;
  int64_t num_out;
  std::tie(inverse_indices, counts, unique_indices, num_out) = compute_unique(
    policy, indices_data, num_inp, indices,
    return_inverse, return_counts, return_indices, options,
    [=] __device__ (int64_t a, int64_t b) -> bool {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = input_flat_ptr[i + a * n];
        scalar_t rhs = input_flat_ptr[i + b * n];
        if (lhs != rhs) {
          return false;
        }
      }
      return true;
    },
    [=] __device__ (int64_t a, int64_t b) -> int64_t {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = input_flat_ptr[i + a * n];
        scalar_t rhs = input_flat_ptr[i + b * n];
        if (lhs != rhs) {
          return 1;
        }
      }
      return 0;
    }
  );
  indices.resize_(num_out);

  return std::tuple<Tensor, Tensor, Tensor, Tensor>(self.index_select(dim, indices), inverse_indices, counts, unique_indices);
}

} // namespace


std::tuple<Tensor, Tensor>
_unique_cuda(const Tensor& self, const bool sorted, const bool return_inverse) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    Tensor output, inverse;
    std::tie(output, inverse, std::ignore, std::ignore) = unique_cuda_template<scalar_t>(self, false, return_inverse, false, false);
    return std::make_tuple(output, inverse);
  });
}

std::tuple<Tensor, Tensor, Tensor>
_unique2_cuda(const Tensor& self, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    Tensor output, inverse, counts;
    std::tie(output, inverse, counts, std::ignore) = unique_cuda_template<scalar_t>(self, false, return_inverse, return_counts, false);
    return std::make_tuple(output, inverse, counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_cuda(const Tensor& self, const int64_t dim, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    Tensor output, inverse, counts;
    std::tie(output, inverse, counts, std::ignore) = unique_dim_cuda_template<scalar_t>(self, dim, false, return_inverse, return_counts, false);
    return std::make_tuple(output, inverse, counts);
    //return unique_dim_cuda_template<scalar_t>(self, dim, false, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_consecutive_cuda(const Tensor& self, const int64_t dim, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    Tensor output, inverse, counts;
    std::tie(output, inverse, counts, std::ignore) = unique_dim_cuda_template<scalar_t>(self, dim, true, return_inverse, return_counts, false);
    return std::make_tuple(output, inverse, counts);
    //return unique_dim_cuda_template<scalar_t>(self, dim, true, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_consecutive_cuda(const Tensor& self, const bool return_inverse, const bool return_counts, c10::optional<int64_t> dim) {
  if (!dim.has_value()) {
    return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
      // The current CUDA implementation of unique always sort due to the
      // lack of hashtable implementation in thrust
      Tensor output, inverse, counts;
      std::tie(output, inverse, counts, std::ignore) = unique_cuda_template<scalar_t>(self, true, return_inverse, return_counts, false);
      return std::make_tuple(output, inverse, counts);
      //return unique_cuda_template<scalar_t>(self, true, return_inverse, return_counts);
    });
  }
  return unique_dim_consecutive_cuda(self, dim.value(), return_inverse, return_counts);
}

std::tuple<Tensor,Tensor,Tensor,Tensor>
uniq_dim_cuda(const Tensor & self, int64_t dim, bool return_inverse, bool return_index, bool return_counts){
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    Tensor output, inverse, counts, indices;
    std::tie(output, inverse, counts, indices) = unique_dim_cuda_template<scalar_t>(self, dim, false, return_inverse, return_counts, return_index);
    return std::make_tuple(output, inverse, indices, counts);
  });
}

std::tuple<Tensor,Tensor,Tensor,Tensor>
uniq_cuda(const Tensor & self, bool return_inverse, bool return_index, bool return_counts){
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    Tensor output, inverse, counts, indices;
    std::tie(output, inverse, counts, indices) = unique_cuda_template<scalar_t>(self, false, return_inverse, return_counts, return_index);
    return std::make_tuple(output, inverse, indices, counts);
  });
}

}  // namespace native
}  // namespace at
