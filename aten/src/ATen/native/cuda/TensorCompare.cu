#include "hip/hip_runtime.h"
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorCompare.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/cuda/CUDAApplyUtils.cuh>


namespace at { namespace native { namespace {

void where_kernel_impl(TensorIterator &iter, ScalarType condition_type) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBFloat16, kBool, iter.dtype(), "where_cuda", [&] {
    if (condition_type == at::ScalarType::Byte) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (uint8_t cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (bool cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    }
  });
}

void isposinf_kernel_impl(TensorIterator &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isposinf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void isneginf_kernel_impl(TensorIterator &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isneginf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == -std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void isin_default_kernel_gpu(Tensor& out, const Tensor& elements, const Tensor& test_elements, bool invert) {
  std::vector<int64_t> bc_shape(elements.dim(), 1);
  bc_shape.push_back(-1);
  out.copy_(invert ? elements.unsqueeze(-1).ne(test_elements.view(bc_shape)).all(-1)
    : elements.unsqueeze(-1).eq(test_elements.view(bc_shape)).any(-1));
}

} // anonymous namespace


REGISTER_DISPATCH(where_kernel, &where_kernel_impl);
REGISTER_DISPATCH(isposinf_stub, &isposinf_kernel_impl);
REGISTER_DISPATCH(isneginf_stub, &isneginf_kernel_impl);
REGISTER_DISPATCH(isin_default_stub, &isin_default_kernel_gpu);

template <typename scalar_t>
__global__ void _assert_async_cuda_kernel(scalar_t* input) {
  CUDA_KERNEL_ASSERT(input[0] != 0);
}

__global__ void _assert_async_cuda_kernel(c10::complex<float>* input) {
  CUDA_KERNEL_ASSERT(input[0] != c10::complex<float>(0, 0));
}
__global__ void _assert_async_cuda_kernel(c10::complex<double>* input) {
  CUDA_KERNEL_ASSERT(input[0] != c10::complex<double>(0, 0));
}

void _assert_async_cuda(const Tensor& self) {
  auto n = self.numel();
  TORCH_CHECK(n != 0, "Boolean value of Tensor with no values is ambiguous");
  TORCH_CHECK(n < 2, "Boolean value of Tensor with more than one value is ambiguous");
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16, self.scalar_type(), "_assert_async_cuda", [&] {
    _assert_async_cuda_kernel<<<1, 1, 0, stream>>>(self.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

}} // namespace at::native
